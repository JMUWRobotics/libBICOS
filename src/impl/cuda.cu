#include "config.hpp"
#include "cuda.hpp"

#include "impl/cuda/agree.cuh"
#include "impl/cuda/bicos.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <cstdint>
#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>

namespace BICOS::impl::cuda {

dim3 create_grid(dim3 block, cv::Size sz) {
    return dim3(
        cv::cuda::device::divUp(sz.width, block.x),
        cv::cuda::device::divUp(sz.height, block.y)
    );
}

template<typename TInput, typename TDescriptor>
static void match_impl(
    const std::vector<cv::cuda::GpuMat>& _stack0,
    const std::vector<cv::cuda::GpuMat>& _stack1,
    size_t n_images,
    cv::Size sz,
    double nxcorr_threshold,
    std::optional<float> subpixel_step,
    cv::cuda::GpuMat& out,
    cv::cuda::Stream& _stream
) {
    std::vector<cv::cuda::PtrStepSz<TInput>> ptrs_host(2 * n_images);
    cv::cuda::PtrStepSz<TInput>* ptrs_dev;

    for (size_t i = 0; i < n_images; ++i) {
        ptrs_host[i] = _stack0[i];
        ptrs_host[i + n_images] = _stack1[i];
    }

    cudaSafeCall(hipHostRegister(
        ptrs_host.data(),
        2 * n_images * sizeof(cv::cuda::PtrStepSz<TInput>),
        hipHostRegisterReadOnly
    ));
    cudaSafeCall(hipHostGetDevicePointer(&ptrs_dev, ptrs_host.data(), 0));

    auto descr0 = std::make_unique<StepBuf<TDescriptor>>(sz),
         descr1 = std::make_unique<StepBuf<TDescriptor>>(sz);

    StepBuf<TDescriptor> *descr0_dev, *descr1_dev;

    cudaSafeCall(hipHostRegister(descr0.get(), sizeof(StepBuf<TDescriptor>), 0));
    cudaSafeCall(hipHostRegister(descr1.get(), sizeof(StepBuf<TDescriptor>), 0));
    cudaSafeCall(hipHostGetDevicePointer(&descr0_dev, descr0.get(), 0));
    cudaSafeCall(hipHostGetDevicePointer(&descr1_dev, descr1.get(), 0));

    size_t smem_size;
    dim3 block(1024);
    dim3 grid(
        cv::cuda::device::divUp(sz.width, block.x),
        cv::cuda::device::divUp(sz.height, block.y)
    );

    hipStream_t mainstream = cv::cuda::StreamAccessor::getStream(_stream);

    /* descriptor transform */

    hipStream_t substream0, substream1;
    hipStreamCreate(&substream0);
    hipStreamCreate(&substream1);

    hipEvent_t event0, event1;
    hipEventCreate(&event0);
    hipEventCreate(&event1);

    smem_size = 0; //block.x * n_images * sizeof(TInput);

    descriptor_transform_kernel<TInput, TDescriptor>
        <<<grid, block, smem_size, substream0>>>(ptrs_dev, n_images, sz, descr0_dev);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipEventRecord(event0, substream0));
    descriptor_transform_kernel<TInput, TDescriptor>
        <<<grid, block, smem_size, substream1>>>(ptrs_dev + n_images, n_images, sz, descr1_dev);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipEventRecord(event1, substream1));

    cudaSafeCall(hipStreamWaitEvent(mainstream, event0));
    cudaSafeCall(hipStreamWaitEvent(mainstream, event1));

#ifdef BICOS_DEBUG
    hipDeviceSynchronize();
#endif

    /* bicos disparity */

    cv::cuda::GpuMat bicos_disp(sz, CV_16SC1);
    bicos_disp.setTo(INVALID_DISP_<int16_t>);

    smem_size = sz.width * sizeof(TDescriptor);

    cudaSafeCall(hipFuncSetAttribute(reinterpret_cast<const void*>(
        bicos_kernel<TDescriptor>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    ));
    bicos_kernel<TDescriptor>
        <<<grid, block, smem_size, mainstream>>>(descr0_dev, descr1_dev, bicos_disp);
    cudaSafeCall(hipGetLastError());

#ifdef BICOS_DEBUG
    hipDeviceSynchronize();
#endif

    /* nxcorr */

    out.create(sz, cv::DataType<disparity_t>::type);
    out.setTo(INVALID_DISP);

    // smem_size = sz.width * n_images * sizeof(TInput);

    block = dim3(768);
    grid = create_grid(block, sz);

    if (subpixel_step.has_value()) {
        cudaSafeCall(hipDeviceSetLimit(
            hipLimitStackSize,
            1024 + 3 * n_images * (sizeof(TInput) + sizeof(float))
        ));
        agree_subpixel_kernel<TInput><<<grid, block, 0, mainstream>>>(
            bicos_disp,
            ptrs_dev,
            n_images,
            sz,
            nxcorr_threshold,
            subpixel_step.value(),
            out
        );
    } else {
        cudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, 1024 + 2 * n_images * sizeof(TInput)));
        agree_kernel<TInput><<<grid, block, 0, mainstream>>>(
            bicos_disp,
            ptrs_dev,
            n_images,
            sz,
            nxcorr_threshold,
            out
        );
    }

    cudaSafeCall(hipGetLastError());

#ifdef BICOS_DEBUG
    hipDeviceSynchronize();
#endif

    cudaSafeCall(hipHostUnregister(descr1.get()));
    cudaSafeCall(hipHostUnregister(descr0.get()));
    cudaSafeCall(hipHostUnregister(ptrs_host.data()));
}

void match(
    const std::vector<cv::cuda::GpuMat>& _stack0,
    const std::vector<cv::cuda::GpuMat>& _stack1,
    cv::cuda::GpuMat& disparity,
    Config cfg,
    cv::cuda::Stream& stream
) {
    const size_t n_images = _stack0.size();
    const int depth = _stack0.front().depth();
    const cv::Size sz = _stack0.front().size();

    int required_bits = cfg.mode == TransformMode::FULL
        ? throw std::invalid_argument("unimplemented")
        : 4 * n_images - 7;

    switch (required_bits) {
        case 0 ... 32:
            if (depth == CV_8U)
                match_impl<uint8_t, uint32_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            else
                match_impl<uint16_t, uint32_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            break;
        case 33 ... 64:
            if (depth == CV_8U)
                match_impl<uint8_t, uint64_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            else
                match_impl<uint16_t, uint64_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            break;
        case 65 ... 128:
            if (depth == CV_8U)
                match_impl<uint8_t, uint128_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            else
                match_impl<uint16_t, uint128_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            break;
        default:
            throw std::invalid_argument("input stacks too large, exceeding 128 bits");
    }
}

} // namespace BICOS::impl::cuda