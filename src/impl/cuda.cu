/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.hpp"
#include "compat.hpp"
#include "cuda.hpp"

#include "impl/common.hpp"
#include "impl/cuda/agree.cuh"
#include "impl/cuda/bicos.cuh"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>

namespace BICOS::impl::cuda {

template<typename TInput, typename TDescriptor>
static void match_impl(
    const std::vector<cv::cuda::GpuMat>& _stack0,
    const std::vector<cv::cuda::GpuMat>& _stack1,
    size_t n_images,
    cv::Size sz,
    double nxcorr_threshold,
    Precision precision,
    TransformMode mode,
    std::optional<float> subpixel_step,
    std::optional<double> min_var,
    std::optional<int> lr_max_diff,
    cv::cuda::GpuMat& out,
    cv::cuda::Stream& _stream
) {
    std::vector<cv::cuda::PtrStepSz<TInput>> ptrs_host(2 * n_images);

    for (size_t i = 0; i < n_images; ++i) {
        ptrs_host[i] = _stack0[i];
        ptrs_host[i + n_images] = _stack1[i];
    }

    StepBuf<TDescriptor> descr0(sz), descr1(sz);

    hipStream_t mainstream = cv::cuda::StreamAccessor::getStream(_stream);

    /* descriptor transform */

    hipStream_t substream0, substream1;
    assertCudaSuccess(hipStreamCreate(&substream0));
    assertCudaSuccess(hipStreamCreate(&substream1));

    hipEvent_t event0, event1;
    assertCudaSuccess(hipEventCreate(&event0));
    assertCudaSuccess(hipEventCreate(&event1));

    RegisteredPtr ptrs_dev(ptrs_host.data(), 2 * n_images, true);
    RegisteredPtr descr0_dev(&descr0), descr1_dev(&descr1);

    dim3 block, grid;

    if (mode == TransformMode::LIMITED) {
        block = max_blocksize(transform_limited_kernel<TInput, TDescriptor>);
        grid = create_grid(block, sz);

        transform_limited_kernel<TInput, TDescriptor>
            <<<grid, block, 0, substream0>>>(ptrs_dev, n_images, sz, descr0_dev);
    } else {
        block = max_blocksize(transform_full_kernel<TInput, TDescriptor>);
        grid = create_grid(block, sz);

        transform_full_kernel<TInput, TDescriptor>
            <<<grid, block, 0, substream0>>>(ptrs_dev, n_images, sz, descr0_dev);
    }

    assertCudaSuccess(hipGetLastError());
    assertCudaSuccess(hipEventRecord(event0, substream0));

    if (mode == TransformMode::LIMITED)
        transform_limited_kernel<TInput, TDescriptor>
            <<<grid, block, 0, substream1>>>(ptrs_dev + n_images, n_images, sz, descr1_dev);
    else
        transform_full_kernel<TInput, TDescriptor>
            <<<grid, block, 0, substream1>>>(ptrs_dev + n_images, n_images, sz, descr1_dev);

    assertCudaSuccess(hipGetLastError());
    assertCudaSuccess(hipEventRecord(event1, substream1));

    assertCudaSuccess(hipStreamWaitEvent(mainstream, event0));
    assertCudaSuccess(hipStreamWaitEvent(mainstream, event1));

    /* bicos disparity */

    cv::cuda::GpuMat bicos_disp(sz, cv::DataType<int16_t>::type);
    bicos_disp.setTo(INVALID_DISP_<int16_t>, _stream);

    auto kernel = lr_max_diff.has_value()
        ? bicos_kernel_smem<TDescriptor, BICOSVariant::WITH_REVERSE>
        : bicos_kernel_smem<TDescriptor, BICOSVariant::DEFAULT>;

    size_t smem_size = sz.width * sizeof(TDescriptor);
    bool bicos_smem_fits = hipSuccess
        == hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    hipGetLastError(); // clear potential error from previous call to hipFuncSetAttribute

    if (bicos_smem_fits) {
        block = max_blocksize(kernel, smem_size);
        grid = create_grid(block, sz);

        kernel<<<grid, block, smem_size, mainstream>>>(descr0_dev, descr1_dev, lr_max_diff.value_or(-1), bicos_disp);
    } else {
        kernel = lr_max_diff.has_value()
            ? bicos_kernel<TDescriptor, BICOSVariant::WITH_REVERSE>
            : bicos_kernel<TDescriptor, BICOSVariant::DEFAULT>;

        block = max_blocksize(kernel);
        grid = create_grid(block, sz);

        kernel<<<grid, block, 0, mainstream>>>(descr0_dev, descr1_dev, lr_max_diff.value_or(-1), bicos_disp);
    }
    assertCudaSuccess(hipGetLastError());

    /* nxcorr */

    out.create(sz, cv::DataType<disparity_t>::type);
    out.setTo(INVALID_DISP, _stream);

    // clang-format off

    switch (precision) {
    case Precision::SINGLE: {

        static agree_kernel_t<TInput, float> lut[2][2] = {
            { agree_kernel<TInput, float, NXCVariant::PLAIN>, agree_kernel<TInput, float, NXCVariant::MINVAR> },
            { agree_subpixel_kernel<TInput, float, NXCVariant::PLAIN>, agree_subpixel_kernel<TInput, float, NXCVariant::MINVAR> }
        };

        auto kernel = lut[subpixel_step.has_value()][min_var.has_value()];

        block = max_blocksize(kernel);
        grid = create_grid(block, sz);

        kernel<<<grid, block, 0, mainstream>>>(
        bicos_disp, ptrs_dev, n_images, nxcorr_threshold, subpixel_step.value_or(0.0f), n_images * min_var.value_or(0.0f), out);

    } break;
    case Precision::DOUBLE: {

        static agree_kernel_t<TInput, double> lut[2][2] = {
            { agree_kernel<TInput, double, NXCVariant::PLAIN>, agree_kernel<TInput, double, NXCVariant::MINVAR> },
            { agree_subpixel_kernel<TInput, double, NXCVariant::PLAIN>, agree_subpixel_kernel<TInput, double, NXCVariant::MINVAR> }
        };

        auto kernel = lut[subpixel_step.has_value()][min_var.has_value()];

        block = max_blocksize(kernel);
        grid = create_grid(block, sz);

        kernel<<<grid, block, 0, mainstream>>>(
        bicos_disp, ptrs_dev, n_images, nxcorr_threshold, subpixel_step.value_or(0.0f), n_images * min_var.value_or(0.0), out);

    } break;
    }

    // clang-format on

    assertCudaSuccess(hipGetLastError());
}

void match(
    const std::vector<cv::cuda::GpuMat>& _stack0,
    const std::vector<cv::cuda::GpuMat>& _stack1,
    cv::cuda::GpuMat& disparity,
    Config cfg,
    cv::cuda::Stream& stream
) {
    const size_t n = _stack0.size();
    const int depth = _stack0.front().depth();
    const cv::Size sz = _stack0.front().size();

    // clang-format off

    int required_bits = cfg.mode == TransformMode::FULL
        ? n * n - 2 * n + 3
        : 4 * n - 7;

    std::optional<int> lr_max_diff = std::nullopt;
    if (std::holds_alternative<Variant::WithReverse>(cfg.variant))
        lr_max_diff = std::get<Variant::WithReverse>(cfg.variant).max_lr_diff;

    switch (required_bits) {
        case 0 ... 32:
            if (depth == CV_8U)
                match_impl<uint8_t, uint32_t>(_stack0, _stack1, n, sz, cfg.nxcorr_thresh, cfg.precision, cfg.mode, cfg.subpixel_step, cfg.min_variance, lr_max_diff, disparity, stream);
            else
                match_impl<uint16_t, uint32_t>(_stack0, _stack1, n, sz, cfg.nxcorr_thresh, cfg.precision, cfg.mode, cfg.subpixel_step, cfg.min_variance, lr_max_diff, disparity, stream);
            break;
        case 33 ... 64:
            if (depth == CV_8U)
                match_impl<uint8_t, uint64_t>(_stack0, _stack1, n, sz, cfg.nxcorr_thresh, cfg.precision, cfg.mode, cfg.subpixel_step, cfg.min_variance, lr_max_diff, disparity, stream);
            else
                match_impl<uint16_t, uint64_t>(_stack0, _stack1, n, sz, cfg.nxcorr_thresh, cfg.precision, cfg.mode, cfg.subpixel_step, cfg.min_variance, lr_max_diff, disparity, stream);
            break;
#ifdef BICOS_CUDA_HAS_UINT128
        case 65 ... 128:
            if (depth == CV_8U)
                match_impl<uint8_t, uint128_t>(_stack0, _stack1, n, sz, cfg.nxcorr_thresh, cfg.precision, cfg.mode, cfg.subpixel_step, cfg.min_variance, lr_max_diff, disparity, stream);
            else
                match_impl<uint16_t, uint128_t>(_stack0, _stack1, n, sz, cfg.nxcorr_thresh, cfg.precision, cfg.mode, cfg.subpixel_step, cfg.min_variance, lr_max_diff, disparity, stream);
            break;
#endif
        default:
            throw std::invalid_argument(BICOS::format("input stacks too large, would require {} bits", required_bits));
    }

    // clang-format on
}

} // namespace BICOS::impl::cuda