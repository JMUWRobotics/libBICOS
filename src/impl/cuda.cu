#include "config.hpp"
#include "cuda.hpp"

#include "impl/cuda/agree.cuh"
#include "impl/cuda/bicos.cuh"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <cstdint>
#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda_stream_accessor.hpp>

namespace BICOS::impl::cuda {

dim3 create_grid(dim3 block, cv::Size sz) {
    return dim3(
        cv::cuda::device::divUp(sz.width, block.x),
        cv::cuda::device::divUp(sz.height, block.y)
    );
}

template<typename TInput, typename TDescriptor>
static void match_impl(
    const std::vector<cv::cuda::GpuMat>& _stack0,
    const std::vector<cv::cuda::GpuMat>& _stack1,
    size_t n_images,
    cv::Size sz,
    double nxcorr_threshold,
    std::optional<float> subpixel_step,
    cv::cuda::GpuMat& out,
    cv::cuda::Stream& _stream
) {
    std::vector<cv::cuda::PtrStepSz<TInput>> ptrs_host(2 * n_images);

    for (size_t i = 0; i < n_images; ++i) {
        ptrs_host[i] = _stack0[i];
        ptrs_host[i + n_images] = _stack1[i];
    }

    StepBuf<TDescriptor> descr0(sz), descr1(sz);

    size_t smem_size;
    dim3 block(1024);
    dim3 grid = create_grid(block, sz);

    hipStream_t mainstream = cv::cuda::StreamAccessor::getStream(_stream);

    /* descriptor transform */

    hipStream_t substream0, substream1;
    hipStreamCreate(&substream0);
    hipStreamCreate(&substream1);

    hipEvent_t event0, event1;
    hipEventCreate(&event0);
    hipEventCreate(&event1);

    RegisteredPtr ptrs_dev(ptrs_host.data(), 2 * n_images, true);
    RegisteredPtr descr0_dev(&descr0), descr1_dev(&descr1);

    descriptor_transform_kernel<TInput, TDescriptor>
        <<<grid, block, 0, substream0>>>(ptrs_dev, n_images, sz, descr0_dev);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipEventRecord(event0, substream0));

    descriptor_transform_kernel<TInput, TDescriptor>
        <<<grid, block, 0, substream1>>>(ptrs_dev + n_images, n_images, sz, descr1_dev);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipEventRecord(event1, substream1));

    cudaSafeCall(hipStreamWaitEvent(mainstream, event0));
    cudaSafeCall(hipStreamWaitEvent(mainstream, event1));

    /* bicos disparity */

    cv::cuda::GpuMat bicos_disp(sz, cv::DataType<int16_t>::type);
    bicos_disp.setTo(INVALID_DISP_<int16_t>);

    smem_size = sz.width * sizeof(TDescriptor);

    cudaSafeCall(hipFuncSetAttribute(reinterpret_cast<const void*>(
        bicos_kernel<TDescriptor>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    ));
    bicos_kernel<TDescriptor>
        <<<grid, block, smem_size, mainstream>>>(descr0_dev, descr1_dev, bicos_disp);
    cudaSafeCall(hipGetLastError());

    /* nxcorr */

    out.create(sz, cv::DataType<disparity_t>::type);
    out.setTo(INVALID_DISP);

    // smem_size = sz.width * n_images * sizeof(TInput);

    block = dim3(768);
    grid = create_grid(block, sz);

    if (subpixel_step.has_value()) {
        cudaSafeCall(hipDeviceSetLimit(
            hipLimitStackSize,
            1024 + 3 * n_images * (sizeof(TInput) + sizeof(float))
        ));
        agree_subpixel_kernel<TInput><<<grid, block, 0, mainstream>>>(
            bicos_disp,
            ptrs_dev,
            n_images,
            nxcorr_threshold,
            subpixel_step.value(),
            out
        );
    } else {
        cudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, 1024 + 2 * n_images * sizeof(TInput)));
        agree_kernel<TInput>
            <<<grid, block, 0, mainstream>>>(bicos_disp, ptrs_dev, n_images, nxcorr_threshold, out);
    }

    cudaSafeCall(hipGetLastError());
}

void match(
    const std::vector<cv::cuda::GpuMat>& _stack0,
    const std::vector<cv::cuda::GpuMat>& _stack1,
    cv::cuda::GpuMat& disparity,
    Config cfg,
    cv::cuda::Stream& stream
) {
    const size_t n_images = _stack0.size();
    const int depth = _stack0.front().depth();
    const cv::Size sz = _stack0.front().size();

    int required_bits = cfg.mode == TransformMode::FULL
        ? throw std::invalid_argument("unimplemented")
        : 4 * n_images - 7;

    switch (required_bits) {
        case 0 ... 32:
            if (depth == CV_8U)
                match_impl<uint8_t, uint32_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            else
                match_impl<uint16_t, uint32_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            break;
        case 33 ... 64:
            if (depth == CV_8U)
                match_impl<uint8_t, uint64_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            else
                match_impl<uint16_t, uint64_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            break;
        case 65 ... 128:
            if (depth == CV_8U)
                match_impl<uint8_t, uint128_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            else
                match_impl<uint16_t, uint128_t>(
                    _stack0,
                    _stack1,
                    n_images,
                    sz,
                    cfg.nxcorr_thresh,
                    cfg.subpixel_step,
                    disparity,
                    stream
                );
            break;
        default:
            throw std::invalid_argument("input stacks too large, exceeding 128 bits");
    }
}

} // namespace BICOS::impl::cuda