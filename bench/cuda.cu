#include "hip/hip_runtime.h"
/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include <benchmark/benchmark.h>

#include <opencv2/core/cuda.hpp>
#include <random>

#include "common.hpp"
#include "cuda.hpp"
#include "fileutils.hpp"
#include "impl/cuda/agree.cuh"
#include "impl/cuda/bicos.cuh"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"
#include "opencv2/core.hpp"
#include "opencv2/core/traits.hpp"
#include "stepbuf.hpp"

using namespace BICOS;
using namespace impl;

constexpr int seed = 0x600DF00D;

constexpr double thresh = 0.9;
constexpr double minvar = 10.0;
constexpr float step = 0.25;
static const cv::Size size(3300, 2200);

template<typename TPrecision, cuda::NXCVariant VARIANT>
__global__ void nxcorr_kernel(const uint8_t* a, const uint8_t* b, size_t n, TPrecision* out, TPrecision minvar) {
    if constexpr (std::is_same_v<TPrecision, float>)
        *out = cuda::nxcorrf<VARIANT>(a, b, n, minvar);
    else
        *out = cuda::nxcorrd<VARIANT>(a, b, n, minvar);
}

template<typename TPrecision, cuda::NXCVariant VARIANT>
void bench_nxcorr_subroutine(benchmark::State& state) {
    uint8_t _a[50], _b[50], *a, *b;
    TPrecision minvar = 100;

    for (size_t i = 0; i < sizeof(_a); ++i) {
        _a[i] = rand();
        _b[i] = rand();
    }

    hipMalloc(&a, sizeof(_a));
    hipMalloc(&b, sizeof(_b));

    hipMemcpy(a, _a, sizeof(_a), hipMemcpyHostToDevice);
    hipMemcpy(b, _b, sizeof(_b), hipMemcpyHostToDevice);

    TPrecision* out;
    hipMalloc(&out, 1);

    for (auto _: state) {
        nxcorr_kernel<TPrecision, VARIANT><<<1, 1>>>(a, b, sizeof(_a), out, minvar);
        hipDeviceSynchronize();
    }
}

template<typename TInput>
void bench_agree_kernel(benchmark::State& state) {
    cv::setRNGSeed(seed);
    const int n = 10;

    cv::Mat_<int16_t> randdisp(size);
    cv::randu(randdisp, -1, size.width);
    cv::cuda::GpuMat randdisp_dev(randdisp);

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::cuda::PtrStepSz<TInput>> devinput;

    for (int i = 0; i < 2 * n; ++i) {
        cv::Mat_<TInput> randmat(size);
        cv::randu(randmat, 0, std::numeric_limits<TInput>::max());

        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }

    cuda::RegisteredPtr devptr(devinput.data(), 2 * n, true);

    cv::cuda::GpuMat out(size, cv::DataType<disparity_t>::type);

    const dim3 block = cuda::max_blocksize(cuda::agree_kernel<TInput, double, cuda::NXCVariant::MINVAR>);
    const dim3 grid = create_grid(block, size);

    for (auto _: state) {
        cuda::agree_kernel<TInput, double, cuda::NXCVariant::MINVAR>
            <<<grid, block>>>(randdisp_dev, devptr, n, thresh, 0.0, minvar, out);
        hipDeviceSynchronize();
    }

    assertCudaSuccess(hipGetLastError());
}

template<typename TInput>
void bench_agree_subpixel_kernel(benchmark::State& state) {
    cv::setRNGSeed(seed);
    const int n = 10;

    cv::Mat_<int16_t> randdisp(size);
    cv::randu(randdisp, -1, size.width);
    cv::cuda::GpuMat randdisp_dev(randdisp);

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::cuda::PtrStepSz<TInput>> devinput;

    for (int i = 0; i < 2 * n; ++i) {
        cv::Mat_<TInput> randmat(size);
        cv::randu(randmat, 0, std::numeric_limits<TInput>::max());

        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }

    cuda::RegisteredPtr devptr(devinput.data(), 2 * n, true);

    cv::cuda::GpuMat out(size, cv::DataType<disparity_t>::type);

    const dim3 block = cuda::max_blocksize(cuda::agree_subpixel_kernel<TInput, double, cuda::NXCVariant::MINVAR>);
    const dim3 grid = create_grid(block, size);

    for (auto _: state) {
        cuda::agree_subpixel_kernel<TInput, double, cuda::NXCVariant::MINVAR>
            <<<grid, block>>>(randdisp_dev, devptr, n, thresh, step, minvar, out);
        hipDeviceSynchronize();
    }

    assertCudaSuccess(hipGetLastError());
}

template<typename TInput>
void bench_agree_subpixel_kernel_smem(benchmark::State& state) {
    cv::setRNGSeed(seed);
    const int n = 10;

    cv::Mat_<int16_t> randdisp(size);
    cv::randu(randdisp, -1, size.width);
    cv::cuda::GpuMat randdisp_dev(randdisp);

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::cuda::PtrStepSz<TInput>> devinput;

    for (int i = 0; i < 2 * n; ++i) {
        cv::Mat_<TInput> randmat(size);
        cv::randu(randmat, 0, std::numeric_limits<TInput>::max());

        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }

    cuda::RegisteredPtr devptr(devinput.data(), 2 * n, true);

    cv::cuda::GpuMat out(size, cv::DataType<disparity_t>::type);

    size_t smem_size = size.width * n * sizeof(TInput);

    bool smem_fits = hipSuccess == hipFuncSetAttribute(reinterpret_cast<const void*>(
        cuda::agree_subpixel_kernel_smem<TInput), double, cuda::NXCVariant::MINVAR>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    );

    if (!smem_fits) {
        state.SkipWithMessage("smem too small");
        return;
    }

    const dim3 block = cuda::max_blocksize(cuda::agree_subpixel_kernel_smem<TInput, double, cuda::NXCVariant::MINVAR>, smem_size);
    const dim3 grid = create_grid(block, size);

    for (auto _: state) {
        cuda::agree_subpixel_kernel_smem<TInput, double, cuda::NXCVariant::MINVAR>
            <<<grid, block, smem_size>>>(randdisp_dev, devptr, n, thresh, step, minvar, out);
        hipDeviceSynchronize();
    }

    assertCudaSuccess(hipGetLastError());
}

template<typename T>
void randomize_seeded(cpu::StepBuf<T>& sb) {
    static thread_local std::independent_bits_engine<std::default_random_engine, CHAR_BIT, uint8_t>
        ibe((uint8_t)seed);

    T* p = sb.row(0);

    std::generate(p, p + sb.size().area(), ibe);
}

template<typename TDescriptor>
void bench_bicos_kernel(benchmark::State& state) {
    cv::setRNGSeed(seed);

    cpu::StepBuf<TDescriptor> ld(size), rd(size);

    randomize_seeded(ld);
    randomize_seeded(rd);

    cuda::StepBuf<TDescriptor> ld_dev(ld), rd_dev(rd);

    cuda::RegisteredPtr lptr(&ld_dev, 1, true), rptr(&rd_dev, 1, true);

    cv::cuda::GpuMat out(size, cv::DataType<int16_t>::type);

    const dim3 block = cuda::max_blocksize(cuda::bicos_kernel<TDescriptor>);
    const dim3 grid = create_grid(block, size);

    for (auto _: state) {
        cuda::bicos_kernel<TDescriptor><<<grid, block>>>(lptr, rptr, out);
        hipDeviceSynchronize();
    }

    assertCudaSuccess(hipGetLastError());
}

template<typename TDescriptor>
void bench_bicos_kernel_smem(benchmark::State& state) {
    cv::setRNGSeed(seed);

    cpu::StepBuf<TDescriptor> ld(size), rd(size);

    randomize_seeded(ld);
    randomize_seeded(rd);

    cuda::StepBuf<TDescriptor> ld_dev(ld), rd_dev(rd);

    cuda::RegisteredPtr lptr(&ld_dev, 1, true), rptr(&rd_dev, 1, true);

    cv::cuda::GpuMat out(size, cv::DataType<int16_t>::type);

    size_t smem_size = size.width * sizeof(TDescriptor);

    bool smem_fits = hipSuccess == hipFuncSetAttribute(reinterpret_cast<const void*>(
        cuda::bicos_kernel_smem<TDescriptor>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    );

    if (!smem_fits) {
        state.SkipWithMessage("smem too small");
        return;
    }

    const dim3 block = cuda::max_blocksize(cuda::bicos_kernel_smem<TDescriptor>, smem_size);
    const dim3 grid = create_grid(block, size);

    for (auto _: state) {
        cuda::bicos_kernel_smem<TDescriptor><<<grid, block, smem_size>>>(lptr, rptr, out);
        hipDeviceSynchronize();
    }

    assertCudaSuccess(hipGetLastError());
}

template<typename TInput, typename TDescriptor, TransformMode mode>
void bench_descriptor_transform_kernel(benchmark::State& state) {
    cv::setRNGSeed(seed);

    int bits = sizeof(TDescriptor) * 8;
    int n = mode == TransformMode::FULL ? int((2 + std::sqrt(4 - 4 * ( 3 - bits ))) / 2.0) : (bits + 7) / 4;

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::cuda::PtrStepSz<TInput>> devinput;

    for (int i = 0; i < n; ++i) {
        cv::Mat_<TInput> randmat(size);
        cv::randu(randmat, 0, std::numeric_limits<TInput>::max());

        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }

    cuda::RegisteredPtr inptr(devinput.data(), n, true);

    cuda::StepBuf<TDescriptor> out(size);
    cuda::RegisteredPtr outptr(&out);

    const dim3 block = cuda::max_blocksize(mode == TransformMode::FULL ? cuda::transform_full_kernel<TInput, TDescriptor> : cuda::transform_limited_kernel<TInput, TDescriptor>);
    const dim3 grid  = create_grid(block, size);

    if constexpr (mode == TransformMode::FULL)
        for (auto _: state)
            cuda::transform_full_kernel<TInput, TDescriptor>
                <<<grid, block>>>(inptr, n, size, outptr);
    else
        for (auto _: state)
            cuda::transform_limited_kernel<TInput, TDescriptor>
                <<<grid, block>>>(inptr, n, size, outptr);

    assertCudaSuccess(hipGetLastError());
}

void bench_integration(benchmark::State& state) {
    std::vector<SequenceEntry> lseq, rseq;
    std::vector<cv::Mat> lhost, rhost;
    std::vector<cv::cuda::GpuMat> ldev, rdev;

    read_sequence(SOURCE_ROOT "/data/left", SOURCE_ROOT "/data/right", lseq, rseq, true);
    sort_sequence_to_stack(lseq, rseq, lhost, rhost);
    matvec_to_gpu(lhost, rhost, ldev, rdev);

    int n = std::min(state.range(0), (int64_t)ldev.size());
    float step = 0.01f * state.range(1);

    ldev.resize(n);
    rdev.resize(n);

    Config c { .nxcorr_thresh = thresh,
               .subpixel_step = step == 0.0f ? std::nullopt : std::optional(step),
               .mode = TransformMode::LIMITED };

    cv::cuda::GpuMat out;
    out.create(ldev.front().size(), cv::DataType<disparity_t>::type);

    for (auto _: state) {
        cuda::match(ldev, rdev, out, c, cv::cuda::Stream::Null());
        hipDeviceSynchronize();
    }
}

BENCHMARK(bench_nxcorr_subroutine<float, cuda::NXCVariant::MINVAR>)
    ->Repetitions(10)
    ->ReportAggregatesOnly(true);
BENCHMARK(bench_nxcorr_subroutine<float, cuda::NXCVariant::PLAIN>)
    ->Repetitions(10)
    ->ReportAggregatesOnly(true);
BENCHMARK(bench_nxcorr_subroutine<double, cuda::NXCVariant::MINVAR>)
    ->Repetitions(10)
    ->ReportAggregatesOnly(true);
BENCHMARK(bench_nxcorr_subroutine<double, cuda::NXCVariant::PLAIN>)
    ->Repetitions(10)
    ->ReportAggregatesOnly(true);


BENCHMARK(bench_agree_kernel<uint8_t>);
BENCHMARK(bench_agree_kernel<uint16_t>);
BENCHMARK(bench_agree_subpixel_kernel<uint8_t>);
BENCHMARK(bench_agree_subpixel_kernel<uint16_t>);
BENCHMARK(bench_agree_subpixel_kernel_smem<uint8_t>);
BENCHMARK(bench_agree_subpixel_kernel_smem<uint16_t>);

BENCHMARK(bench_bicos_kernel<uint32_t>);
BENCHMARK(bench_bicos_kernel<uint64_t>);
BENCHMARK(bench_bicos_kernel<uint128_t>);
BENCHMARK(bench_bicos_kernel_smem<uint32_t>);
BENCHMARK(bench_bicos_kernel_smem<uint64_t>);
BENCHMARK(bench_bicos_kernel_smem<uint128_t>);

BENCHMARK(bench_descriptor_transform_kernel<uint8_t, uint32_t, TransformMode::LIMITED>);
BENCHMARK(bench_descriptor_transform_kernel<uint16_t, uint32_t, TransformMode::LIMITED>);
BENCHMARK(bench_descriptor_transform_kernel<uint8_t, uint64_t, TransformMode::LIMITED>);
BENCHMARK(bench_descriptor_transform_kernel<uint16_t, uint64_t, TransformMode::LIMITED>);
BENCHMARK(bench_descriptor_transform_kernel<uint8_t, uint128_t, TransformMode::LIMITED>);
BENCHMARK(bench_descriptor_transform_kernel<uint16_t, uint128_t, TransformMode::LIMITED>);

BENCHMARK(bench_descriptor_transform_kernel<uint8_t, uint32_t, TransformMode::FULL>);
BENCHMARK(bench_descriptor_transform_kernel<uint16_t, uint32_t, TransformMode::FULL>);
BENCHMARK(bench_descriptor_transform_kernel<uint8_t, uint64_t, TransformMode::FULL>);
BENCHMARK(bench_descriptor_transform_kernel<uint16_t, uint64_t, TransformMode::FULL>);
BENCHMARK(bench_descriptor_transform_kernel<uint8_t, uint128_t, TransformMode::FULL>);
BENCHMARK(bench_descriptor_transform_kernel<uint16_t, uint128_t, TransformMode::FULL>);

BENCHMARK(bench_integration)
    ->ArgsProduct({
        { 2, 8, 14, 20 }, // n
        { 0, 25, 20, 15, 10 } // step * 100
    });

BENCHMARK_MAIN();
