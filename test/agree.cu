#include "hip/hip_runtime.h"
/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "common.hpp"
#include "impl/cpu/agree.hpp"
#include "impl/cuda/agree.cuh"
#include "impl/cuda/cutil.cuh"
#include "opencv2/core.hpp"
#include "opencv2/core/traits.hpp"
#include "opencv2/core/types.hpp"

#include <limits>
#include <opencv2/core/cuda.hpp>

using namespace BICOS;
using namespace impl;
using namespace test;

int main(void) {
    int n = 10;

    const cv::Size randsize(randint(1024, 4096), randint(512, 2048));

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::Mat> _hostinput;
    std::vector<cv::cuda::PtrStepSz<INPUT_TYPE>> devinput;
    cv::Mat hinput_l, hinput_r;

    dim3 grid, block;

    for (int i = 0; i < 2 * n; ++i) {
        if (i == n) {
            cv::merge(_hostinput, hinput_l);
            _hostinput.clear();
        }
        cv::Mat_<INPUT_TYPE> randmat(randsize);
        cv::randu(randmat, 0, std::numeric_limits<INPUT_TYPE>::max());

        _hostinput.push_back(randmat);
        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }
    cv::merge(_hostinput, hinput_r);

    const cuda::RegisteredPtr devptr(devinput.data(), 2 * n, true);

    cv::Mat_<int16_t> randdisp(randsize);
    cv::randu(randdisp, -1, randsize.width);

    {
        double min, max;
        cv::minMaxIdx(randdisp, &min, &max);
        assert(min == -1.0);
    }

    cv::cuda::GpuMat randdisp_dev;
    randdisp_dev.upload(randdisp);

    double thresh = randreal(-0.9, 0.9);
    double minvar = randreal(0.0, 75.0);

#if TEST_SUBPIXEL

    cv::cuda::GpuMat devout(randsize, cv::DataType<float>::type);
    devout.setTo(INVALID_DISP<float>);
    cv::Mat_<float> hostout(randsize), devout_host;

    auto kernel = cuda::agree_subpixel_kernel<INPUT_TYPE, double, cuda::NXCVariant::MINVAR, false>;

    block = cuda::max_blocksize(kernel);
    grid = create_grid(block, randsize);

    float step = 0.25f;

    kernel<<<grid, block>>>(randdisp_dev, devptr, n, thresh, step, minvar, devout, cv::cuda::PtrStepSz<double>());

    assertCudaSuccess(hipGetLastError());

    cpu::agree_subpixel<INPUT_TYPE>(randdisp, hinput_l, hinput_r, n, thresh, step, minvar, hostout, nullptr);

    assertCudaSuccess(hipDeviceSynchronize());

    devout.download(devout_host);

    double err = maxerr(hostout, devout_host);

    // TODO investigate why agree_subpixel fails on random input data

    std::cout << "max-err: " << err << std::endl;
    if (err > 2.0) {
        return 1;
    }

    return 0;

#else

    cv::Mat_<int16_t> devout_host;

    auto kernel = cuda::agree_kernel<INPUT_TYPE, double, cuda::NXCVariant::MINVAR, false>;

    block = cuda::max_blocksize(kernel);
    grid = create_grid(block, randsize);

    kernel<<<grid, block>>>(randdisp_dev, devptr, n, thresh, minvar, cv::cuda::PtrStepSz<double>());

    assertCudaSuccess(hipGetLastError());

    cpu::agree<INPUT_TYPE>(randdisp, hinput_l, hinput_r, n, thresh, minvar, nullptr);

    assertCudaSuccess(hipDeviceSynchronize());

    randdisp_dev.download(devout_host);

    if (!equals(randdisp, devout_host))
        return 1;

    return 0;

#endif
}