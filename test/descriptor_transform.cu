#include "hip/hip_runtime.h"
#include "config.hpp"
#include "impl/cpu/descriptor_transform.hpp"
#include "impl/cuda/descriptor_transform.cuh"
#include "util.cuh"

#include <opencv2/core/cuda.hpp>
#include <cstdlib>
#include <format>
#include <iostream>
#include <random>

using namespace BICOS;
using namespace impl;
using namespace test;

#define _STR(s) #s
#define STR(s) _STR(s)

dim3 create_grid(dim3 block, cv::Size sz) {
    return dim3(
        cv::cuda::device::divUp(sz.width, block.x),
        cv::cuda::device::divUp(sz.height, block.y)
    );
}

template<typename T>
bool equals(const cpu::StepBuf<T>& a, const cpu::StepBuf<T>& b, cv::Size sz) {
    for (int row = 0; row < sz.height; ++row) {
        for (int col = 0; col < sz.width; ++col) {
            T va = a.row(row)[col], vb = b.row(row)[col];
            if (va != vb) {
                std::cerr << std::format("{} != {} at ({},{})\n", va, vb, col, row);
                return false;
            }
        }
    }

    return true;
}

int rnd(int from, int to) {
    std::random_device dev;
    std::uniform_int_distribution<int> dist(from, to - 1);
    int rnum = dist(dev);
    return rnum;
}

int main(int argc, char const* const* argv) {
    srand(time(NULL));

    cv::Mat hoststack;
    std::vector<cv::Mat_<INPUT_TYPE>> rand_host;
    std::vector<cv::cuda::GpuMat> _rand_dev;
    std::vector<cv::cuda::PtrStepSz<INPUT_TYPE>> rand_dev;

    const cv::Size randsize(rnd(1024, 4096), rnd(512, 2048));

    std::cout << "descriptor transform on " << randsize << " " << STR(INPUT_TYPE) << " " << STR(DESCRIPTOR_TYPE) << std::endl;

    int max_bits = sizeof(DESCRIPTOR_TYPE) * 8;
    size_t n = (max_bits + 7) / 4;

    for (size_t i = 0; i < n; ++i) {
        cv::Mat_<INPUT_TYPE> randmat(randsize);
        cv::randu(randmat, 0, std::numeric_limits<INPUT_TYPE>::max());
        rand_host.push_back(randmat);

        cv::cuda::GpuMat randmat_dev(randmat);
        _rand_dev.push_back(randmat_dev);
        rand_dev.push_back(randmat_dev);
    }

    RegisteredPtr rand_devptr(rand_dev.data(), n, true);

    cuda::StepBuf<DESCRIPTOR_TYPE> gpuout(randsize);
    RegisteredPtr gpuout_devptr(&gpuout);

    dim3 block(1024);
    dim3 grid = create_grid(block, randsize);

    cuda::descriptor_transform_kernel<INPUT_TYPE, DESCRIPTOR_TYPE>
        <<<grid, block>>>(rand_devptr, n, randsize, gpuout_devptr);

    cudaSafeCall(hipGetLastError());

    cv::merge(rand_host, hoststack);

    auto cpuout = cpu::descriptor_transform<INPUT_TYPE, DESCRIPTOR_TYPE>(
        hoststack,
        randsize,
        n,
        TransformMode::LIMITED
    );

    cudaSafeCall(hipDeviceSynchronize());

    cpu::StepBuf<DESCRIPTOR_TYPE> gpuout_host(gpuout);

    if (!equals(*cpuout, gpuout_host, randsize))
        return -1;

    return 0;
}
