#include "hip/hip_runtime.h"
/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "formatable.hpp"
#include "impl/cpu/descriptor_transform.hpp"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <opencv2/core/cuda.hpp>

using namespace BICOS;
using namespace impl;
using namespace test;

#define _STR(s) #s
#define STR(s) _STR(s)

#if !defined INPUT_TYPE || !defined DESCRIPTOR_TYPE || !defined TRANSFORM_LIMITED
#   error "undefined compilation constants"
#endif

int main(void) {
    cv::Mat hoststack;
    std::vector<cv::Mat_<INPUT_TYPE>> rand_host;
    std::vector<cv::cuda::GpuMat> _rand_dev;
    std::vector<cv::cuda::PtrStepSz<INPUT_TYPE>> rand_dev;

    const cv::Size randsize(randint(256, 1028), randint(128, 512));

    fmt::println("limited descriptor transform on {} {} {}", randsize, STR(INPUT_TYPE), STR(DESCRIPTOR_TYPE));

    int max_bits = sizeof(DESCRIPTOR_TYPE) * 8;

#if TRANSFORM_LIMITED
    size_t n = (max_bits + 7) / 4;
#else
    size_t n = size_t((2 + std::sqrt(4 - 4 * ( 3 - max_bits ))) / 2.0);
#endif

    for (size_t i = 0; i < n; ++i) {
        cv::Mat_<INPUT_TYPE> randmat(randsize);
        cv::randu(randmat, 0, std::numeric_limits<INPUT_TYPE>::max());
        rand_host.push_back(randmat);

        cv::cuda::GpuMat randmat_dev(randmat);
        _rand_dev.push_back(randmat_dev);
        rand_dev.push_back(randmat_dev);
    }

    const cuda::RegisteredPtr rand_devptr(rand_dev.data(), n, true);

    cuda::StepBuf<DESCRIPTOR_TYPE> gpuout(randsize);
    cuda::RegisteredPtr gpuout_devptr(&gpuout);

    dim3 grid, block;

#if TRANSFORM_LIMITED

    block = cuda::max_blocksize(cuda::transform_limited_kernel<INPUT_TYPE, DESCRIPTOR_TYPE>);
    grid = create_grid(block, randsize);

    cuda::transform_limited_kernel<INPUT_TYPE, DESCRIPTOR_TYPE>
        <<<grid, block>>>(rand_devptr, n, randsize, gpuout_devptr);

#else

    block = cuda::max_blocksize(cuda::transform_limited_kernel<INPUT_TYPE, DESCRIPTOR_TYPE>);
    grid = create_grid(block, randsize);

    cuda::transform_full_kernel<INPUT_TYPE, DESCRIPTOR_TYPE>
        <<<grid, block>>>(rand_devptr, n, randsize, gpuout_devptr);

#endif

    assertCudaSuccess(hipGetLastError());

    cv::merge(rand_host, hoststack);

#if TRANSFORM_LIMITED

    auto cpuout = cpu::descriptor_transform<INPUT_TYPE, DESCRIPTOR_TYPE, cpu::transform_limited>(
        hoststack,
        randsize,
        n
    );

#else

    auto cpuout = cpu::descriptor_transform<INPUT_TYPE, DESCRIPTOR_TYPE, cpu::transform_full>(
        hoststack,
        randsize,
        n
    );

#endif

    assertCudaSuccess(hipDeviceSynchronize());

    cpu::StepBuf<DESCRIPTOR_TYPE> gpuout_host(gpuout);

    if (!equals(*cpuout, gpuout_host, randsize))
        return 1;

    return 0;
}
