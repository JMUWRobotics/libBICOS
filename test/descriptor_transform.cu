#include "hip/hip_runtime.h"
/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "common.hpp"
#include "impl/cpu/descriptor_transform.hpp"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <iostream>
#include <opencv2/core/cuda.hpp>

using namespace BICOS;
using namespace impl;
using namespace test;

#define _STR(s) #s
#define STR(s) _STR(s)

int main(void) {
    cv::Mat hoststack;
    std::vector<cv::Mat_<INPUT_TYPE>> rand_host;
    std::vector<cv::cuda::GpuMat> _rand_dev;
    std::vector<cv::cuda::PtrStepSz<INPUT_TYPE>> rand_dev;

    const cv::Size randsize(randint(256, 1028), randint(128, 512));

    std::cout << "descriptor transform on " << randsize << " " << STR(INPUT_TYPE) << " "
              << STR(DESCRIPTOR_TYPE) << std::endl;

    int max_bits = sizeof(DESCRIPTOR_TYPE) * 8;
    size_t n = (max_bits + 7) / 4;

    for (size_t i = 0; i < n; ++i) {
        cv::Mat_<INPUT_TYPE> randmat(randsize);
        cv::randu(randmat, 0, std::numeric_limits<INPUT_TYPE>::max());
        rand_host.push_back(randmat);

        cv::cuda::GpuMat randmat_dev(randmat);
        _rand_dev.push_back(randmat_dev);
        rand_dev.push_back(randmat_dev);
    }

    cuda::RegisteredPtr rand_devptr(rand_dev.data(), n, true);

    cuda::StepBuf<DESCRIPTOR_TYPE> gpuout(randsize);
    cuda::RegisteredPtr gpuout_devptr(&gpuout);

    const dim3 block(1024);
    const dim3 grid = create_grid(block, randsize);

    cuda::descriptor_transform_kernel<INPUT_TYPE, DESCRIPTOR_TYPE>
        <<<grid, block>>>(rand_devptr, n, randsize, gpuout_devptr);

    assertCudaSuccess(hipGetLastError());

    cv::merge(rand_host, hoststack);

    auto cpuout = cpu::descriptor_transform<INPUT_TYPE, DESCRIPTOR_TYPE>(
        hoststack,
        randsize,
        n,
        TransformMode::LIMITED
    );

    assertCudaSuccess(hipDeviceSynchronize());

    cpu::StepBuf<DESCRIPTOR_TYPE> gpuout_host(gpuout);

    if (!equals(*cpuout, gpuout_host, randsize))
        return 1;

    return 0;
}
