/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "common.hpp"
#include "fileutils.hpp"
#include "impl/cpu/bicos.hpp"
#include "impl/cpu/descriptor_transform.hpp"
#include "impl/cuda/bicos.cuh"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <opencv2/core/cuda_types.hpp>

using namespace BICOS;
using namespace test;
using namespace impl;

int main(int argc, char const* const* argv) {
    std::vector<SequenceEntry> lseq, rseq;
    std::vector<cv::Mat> lhost, rhost;
    std::vector<cv::cuda::GpuMat> _ldev, _rdev;
    std::vector<cv::cuda::PtrStepSz<uint8_t>> dev;

    read_sequence(argv[1], argv[2], lseq, rseq, true);
    sort_sequence_to_stack(lseq, rseq, lhost, rhost);
    matvec_to_gpu(lhost, rhost, _ldev, _rdev);

    const cv::Size sz = lhost.front().size();
    const size_t n = lhost.size();

    for (size_t i = 0; i < n; ++i) {
        dev.push_back(_ldev[i]);
    }
    for (size_t i = 0; i < n; ++i) {
        dev.push_back(_rdev[i]);
    }

    cv::Mat_<int16_t> raw_gpu_host, raw_host;

    cuda::RegisteredPtr dptr(dev.data(), 2 * n, true);

    dim3 grid, block;

    impl::cuda::StepBuf<uint128_t> lddev(sz), rddev(sz);

    cuda::RegisteredPtr ldptr(&lddev), rdptr(&rddev);

    hipStream_t lstream, rstream, mainstream;
    hipStreamCreate(&lstream);
    hipStreamCreate(&rstream);
    hipStreamCreate(&mainstream);

    hipEvent_t ldescev, rdescev;
    hipEventCreate(&ldescev);
    hipEventCreate(&rdescev);

    block = impl::cuda::max_blocksize(impl::cuda::transform_limited_kernel<uint8_t, uint128_t>);
    grid = create_grid(block, sz);

    impl::cuda::transform_limited_kernel<uint8_t, uint128_t>
        <<<grid, block, 0, lstream>>>(dptr, n, sz, ldptr);
    impl::cuda::transform_limited_kernel<uint8_t, uint128_t>
        <<<grid, block, 0, rstream>>>(dptr + n, n, sz, rdptr);

    assertCudaSuccess(hipGetLastError());

    hipEventRecord(ldescev, lstream);
    hipEventRecord(rdescev, rstream);

    hipStreamWaitEvent(mainstream, ldescev);
    hipStreamWaitEvent(mainstream, rdescev);

    cv::cuda::GpuMat raw_gpu(sz, cv::DataType<int16_t>::type);
    raw_gpu.setTo(INVALID_DISP_<int16_t>);

    block = impl::cuda::max_blocksize(impl::cuda::bicos_kernel<uint128_t>);
    grid = create_grid(block, sz);

    impl::cuda::bicos_kernel<uint128_t>
        <<<grid, block, 0, mainstream>>>(ldptr, rdptr, raw_gpu);

    assertCudaSuccess(hipGetLastError());

    cv::Mat lhin, rhin;

    cv::merge(lhost, lhin);
    cv::merge(rhost, rhin);

    auto ldhost = impl::cpu::descriptor_transform<uint8_t, uint128_t, impl::cpu::transform_limited>(
             lhin,
             sz,
             n
         ),
         rdhost = impl::cpu::descriptor_transform<uint8_t, uint128_t, impl::cpu::transform_limited>(
             rhin,
             sz,
             n
         );

    raw_host = impl::cpu::bicos(ldhost, rdhost, sz);

    hipStreamSynchronize(mainstream);

    raw_gpu.download(raw_gpu_host);

    if (!equals(raw_host, raw_gpu_host))
        return 1;

    return 0;
}
