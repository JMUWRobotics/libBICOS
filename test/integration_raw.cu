/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024-2025  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "common.hpp"
#include "fileutils.hpp"
#include "impl/cpu/bicos.hpp"
#include "impl/cpu/descriptor_transform.hpp"
#include "impl/cuda/bicos.cuh"
#include "impl/cuda/cutil.cuh"
#include "impl/cuda/descriptor_transform.cuh"

#include <opencv2/core/cuda_types.hpp>

using namespace BICOS;
using namespace test;
using namespace impl;

#ifdef BICOS_CUDA_HAS_UINT128

int main(int argc, char const* const* argv) {
    std::vector<SequenceEntry> lseq, rseq;
    std::vector<cv::Mat> lhost, rhost;
    std::vector<cv::cuda::GpuMat> _ldev, _rdev;
    std::vector<cuda::GpuMatHeader> dev;

    constexpr size_t n = 22;

    read_sequence(argv[1], argv[2], lseq, rseq, true);
    lseq.resize(n);
    rseq.resize(n);
    sort_sequence_to_stack(lseq, rseq, lhost, rhost);
    matvec_to_gpu(lhost, rhost, _ldev, _rdev);

    const cv::Size sz = lhost.front().size();

    for (size_t i = 0; i < n; ++i) {
        dev.push_back(_ldev[i]);
    }
    for (size_t i = 0; i < n; ++i) {
        dev.push_back(_rdev[i]);
    }

    cv::Mat_<int16_t> raw_gpu_host, raw_host;

    const cuda::RegisteredPtr dptr(dev.data(), 2 * n, true);

    dim3 grid, block;

    cuda::StepBuf<uint128_t> lddev(sz), rddev(sz);

    cuda::RegisteredPtr ldptr(&lddev), rdptr(&rddev);

    hipStream_t lstream, rstream, mainstream;
    hipStreamCreate(&lstream);
    hipStreamCreate(&rstream);
    hipStreamCreate(&mainstream);

    hipEvent_t ldescev, rdescev;
    hipEventCreate(&ldescev);
    hipEventCreate(&rdescev);

    block = cuda::max_blocksize(cuda::transform_limited_kernel<uint8_t, uint128_t, n>);
    grid = create_grid(block, sz);

    cuda::transform_limited_kernel<uint8_t, uint128_t, n>
        <<<grid, block, 0, lstream>>>(dptr, n, sz, ldptr);
    cuda::transform_limited_kernel<uint8_t, uint128_t, n>
        <<<grid, block, 0, rstream>>>(dptr + n, n, sz, rdptr);

    assertCudaSuccess(hipGetLastError());

    hipEventRecord(ldescev, lstream);
    hipEventRecord(rdescev, rstream);

    hipStreamWaitEvent(mainstream, ldescev);
    hipStreamWaitEvent(mainstream, rdescev);

    cv::cuda::GpuMat raw_gpu(sz, cv::DataType<int16_t>::type);
    raw_gpu.setTo(INVALID_DISP<int16_t>);

    block = cuda::max_blocksize(cuda::bicos_kernel<uint128_t, BICOSFLAGS>);
    grid = create_grid(block, sz);

    // clang-format off

    cuda::bicos_kernel<uint128_t, BICOSFLAGS>
        <<<grid, block, 0, mainstream>>>(ldptr, rdptr, 3, raw_gpu);

    assertCudaSuccess(hipGetLastError());

    cv::Mat lhin, rhin;

    cv::merge(lhost, lhin);
    cv::merge(rhost, rhin);

    auto ldhost = cpu::descriptor_transform<uint8_t, uint128_t, impl::cpu::transform_limited>(lhin, sz, n),
         rdhost = cpu::descriptor_transform<uint8_t, uint128_t, impl::cpu::transform_limited>(rhin, sz, n);

    // clang-format on

    cpu::bicos<uint128_t, BICOSFLAGS>(ldhost, rdhost, 3, sz, raw_host);

    hipStreamSynchronize(mainstream);

    raw_gpu.download(raw_gpu_host);

    if (!equals(raw_host, raw_gpu_host))
        return 1;

    return 0;
}

#else

int main(int argc, char const* const* argv) { return EXIT_TEST_SKIP; }

#endif
