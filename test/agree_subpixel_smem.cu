#include "hip/hip_runtime.h"
/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "common.hpp"
#include "impl/cuda/agree.cuh"
#include "impl/cuda/cutil.cuh"

#include <opencv2/core/cuda.hpp>

using namespace BICOS;
using namespace impl;
using namespace test;

int main(void) {
    int n = 15;

    const cv::Size randsize(randint(512, 2048), randint(256, 1024));

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::cuda::PtrStepSz<INPUT_TYPE>> devinput;

    for (int i = 0; i < 2 * n; ++i) {
        cv::Mat_<INPUT_TYPE> randmat(randsize);
        cv::randu(randmat, 0, std::numeric_limits<INPUT_TYPE>::max());

        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }

    RegisteredPtr devptr(devinput.data(), 2 * n, true);

    cv::Mat_<int16_t> randdisp(randsize);
    cv::randu(randdisp, -1, randsize.width);

    cv::cuda::GpuMat randdisp_dev;
    randdisp_dev.upload(randdisp);

    const dim3 block(512);
    const dim3 grid = create_grid(block, randsize);

    double thresh = randreal(-0.9, 0.9);

    cv::cuda::GpuMat devout_gmem(randsize, cv::DataType<disparity_t>::type),
        devout_smem(randsize, cv::DataType<disparity_t>::type);

    devout_gmem.setTo(INVALID_DISP);
    devout_smem.setTo(INVALID_DISP);

    size_t smem_size = randsize.width * n * sizeof(INPUT_TYPE);

    float step = 0.25f;

    cuda::agree_subpixel_kernel<INPUT_TYPE, double, cuda::nxcorrd>
        <<<grid, block>>>(randdisp_dev, devptr, n, thresh, step, devout_gmem);
    assertCudaSuccess(hipGetLastError());

    assertCudaSuccess(hipFuncSetAttribute(reinterpret_cast<const void*>(
        impl::cuda::agree_subpixel_kernel_smem<INPUT_TYPE), double, cuda::nxcorrd>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    ));

    cuda::agree_subpixel_kernel_smem<INPUT_TYPE, double, cuda::nxcorrd>
        <<<grid, block, smem_size>>>(randdisp_dev, devptr, n, thresh, step, devout_smem);
    assertCudaSuccess(hipGetLastError());

    cv::Mat_<disparity_t> gmem, smem;
    devout_gmem.download(gmem);
    devout_smem.download(smem);

    double err = maxerr(gmem, smem);

    std::cout << "max-err: " << err << std::endl;
    if (err > 2.0) {
        return 1;
    }

    return 0;

    return 0;
}
