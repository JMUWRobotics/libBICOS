#include "hip/hip_runtime.h"
/**
 *  libBICOS: binary correspondence search on multishot stereo imagery
 *  Copyright (C) 2024  Robotics Group @ Julius-Maximilian University
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU Lesser General Public License as
 *  published by the Free Software Foundation, either version 3 of the
 *  License, or (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU Lesser General Public License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "common.cuh"
#include "common.hpp"
#include "impl/cuda/agree.cuh"
#include "impl/cuda/cutil.cuh"

#include <opencv2/core/cuda.hpp>

using namespace BICOS;
using namespace impl;
using namespace test;

int main(void) {
    int n = 15;

    const cv::Size randsize(randint(512, 2048), randint(256, 1024));

    std::vector<cv::cuda::GpuMat> _devinput;
    std::vector<cv::cuda::PtrStepSz<INPUT_TYPE>> devinput;

    for (int i = 0; i < 2 * n; ++i) {
        cv::Mat_<INPUT_TYPE> randmat(randsize);
        cv::randu(randmat, 0, std::numeric_limits<INPUT_TYPE>::max());

        cv::cuda::GpuMat randmat_dev(randmat);

        _devinput.push_back(randmat_dev);
        devinput.push_back(randmat_dev);
    }

    const cuda::RegisteredPtr devptr(devinput.data(), 2 * n, true);

    cv::Mat_<int16_t> randdisp(randsize);
    cv::randu(randdisp, -1, randsize.width);

    cv::cuda::GpuMat randdisp_dev;
    randdisp_dev.upload(randdisp);

    dim3 block, grid;

    double thresh = randreal(-0.9, 0.9);
    double minvar = randreal(0.1, 10.0);

    cv::cuda::GpuMat devout_gmem(randsize, cv::DataType<disparity_t>::type),
        devout_smem(randsize, cv::DataType<disparity_t>::type);

    devout_gmem.setTo(INVALID_DISP);
    devout_smem.setTo(INVALID_DISP);

    size_t smem_size = randsize.width * n * sizeof(INPUT_TYPE);

    float step = 0.25f;

    auto kernel = cuda::agree_subpixel_kernel<INPUT_TYPE, double, cuda::NXCVariant::MINVAR>;

    block = cuda::max_blocksize(kernel);
    grid = create_grid(block, randsize);

    kernel<<<grid, block>>>(randdisp_dev, devptr, n, thresh, step, minvar, devout_gmem);
    assertCudaSuccess(hipGetLastError());

    kernel = cuda::agree_subpixel_kernel_smem<INPUT_TYPE, double, cuda::NXCVariant::MINVAR>;

    bool smem_fits = hipSuccess == hipFuncSetAttribute(reinterpret_cast<const void*>(
        kernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_size
    );

    if (!smem_fits)
        return 77; // skip, see https://mesonbuild.com/Unit-tests.html#skipped-tests-and-hard-errors

    block = cuda::max_blocksize(kernel);
    grid = create_grid(block, randsize);

    kernel<<<grid, block, smem_size>>>(randdisp_dev, devptr, n, thresh, step, minvar, devout_smem);
    assertCudaSuccess(hipGetLastError());

    cv::Mat_<disparity_t> gmem, smem;
    devout_gmem.download(gmem);
    devout_smem.download(smem);

    double err = maxerr(gmem, smem);

    std::cout << "max-err: " << err << std::endl;
    if (err > 2.0) {
        return 1;
    }

    return 0;

    return 0;
}
